#include "hip/hip_runtime.h"
#define GLM_FORCE_SWIZZLE
#include <sstream>
#include <hip/hip_runtime.h>
#include "interpolator.h"
#include "kernels.cu"
#include "lfLoader.h"
#include "libs/loadingBar/loadingbar.hpp"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "libs/stb_image_write.h"

class Timer
{
    public:
    Timer()
    {    
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent);
    }
    float stop()
    {
        hipEventRecord(stopEvent);
        hipEventSynchronize(stopEvent);
        float time = 0;
        hipEventElapsedTime(&time, startEvent, stopEvent);
        hipEventDestroy(startEvent);
        hipEventDestroy(stopEvent);
        return time; 
    };
    private:
    hipEvent_t startEvent, stopEvent;
};

Interpolator::Interpolator(std::string inputPath) : input{inputPath}
{
    init();
}

Interpolator::~Interpolator()
{
    hipDeviceReset();
}

void Interpolator::init()
{
    loadGPUData();
    sharedSize = 0;//sizeof(half)*colsRows.x*colsRows.y;
}

int Interpolator::createTextureObject(const uint8_t *data, glm::ivec3 size)
{
    hipChannelFormatDesc channels = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipArray *arr;
    hipMallocArray(&arr, &channels, size.x, size.y);
    hipMemcpy2DToArray(arr, 0, 0, data, size.x*size.z, size.x*size.z, size.y, hipMemcpyHostToDevice);
    
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = arr;
    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeNormalizedFloat;
    hipTextureObject_t texObj{0};
    hipCreateTextureObject(&texObj, &texRes, &texDescr, NULL);
    return texObj;
}

std::pair<int, int*> Interpolator::createSurfaceObject(glm::ivec3 size, const uint8_t *data)
{
    auto arr = loadImageToArray(data, size);
    hipResourceDesc surfRes;
    memset(&surfRes, 0, sizeof(hipResourceDesc));
    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = reinterpret_cast<hipArray*>(arr);
    hipSurfaceObject_t surfObj = 0;
    hipCreateSurfaceObject(&surfObj, &surfRes);
    return {surfObj, arr};
}

int* Interpolator::loadImageToArray(const uint8_t *data, glm::ivec3 size)
{
    hipChannelFormatDesc channels = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned); 
    hipArray *arr;
    hipMallocArray(&arr, &channels, size.x, size.y, hipArraySurfaceLoadStore);
    if(data != nullptr)
        hipMemcpy2DToArray(arr, 0, 0, data, size.x*size.z, size.x*size.z, size.y, hipMemcpyHostToDevice);
    return reinterpret_cast<int*>(arr);
}

void Interpolator::loadGPUData()
{
    LfLoader lfLoader;
    lfLoader.loadData(input);
    colsRows = lfLoader.getColsRows();
    resolution = lfLoader.imageResolution();

    std::cout << "Uploading data to GPU..." << std::endl;
    LoadingBar bar(lfLoader.imageCount()+OUTPUT_SURFACE_COUNT);

    
    std::vector<hipTextureObject_t> textures;
    for(int col=0; col<colsRows.x; col++)
        for(int row=0; row<colsRows.y; row++)
        { 
            textures.push_back(createTextureObject(lfLoader.image({col, row}).data(), resolution)); 
            bar.add();
        }

    hipMalloc(&textureObjectsArr, textures.size()*sizeof(hipTextureObject_t));
    hipMemcpy(textureObjectsArr, textures.data(), textures.size()*sizeof(hipTextureObject_t), hipMemcpyHostToDevice); 
 
    std::vector<hipSurfaceObject_t> surfaces;
/*    for(int col=0; col<colsRows.x; col++)
        for(int row=0; row<colsRows.y; row++)
        {
            auto surface = createSurfaceObject(resolution, lfLoader.image({col, row}).data());
            surfaces.push_back(surface.first);  
            surfaceInputArrays.push_back(surface.second);
            bar.add();
        }
*/
    for(int i=0; i<OUTPUT_SURFACE_COUNT; i++)
    {
        auto surface = createSurfaceObject(resolution);
        surfaces.push_back(surface.first);  
        surfaceOutputArrays.push_back(surface.second);
        bar.add();
    }
    hipMalloc(&surfaceObjectsArr, surfaces.size()*sizeof(hipTextureObject_t));
    hipMemcpy(surfaceObjectsArr, surfaces.data(), surfaces.size()*sizeof(hipSurfaceObject_t), hipMemcpyHostToDevice);
}

void Interpolator::loadGPUConstants(InterpolationParams params)
{
    std::vector<int> intValues(IntConstantIDs::INT_CONSTANTS_COUNT);
    intValues[IntConstantIDs::IMG_RES_X] = resolution.x;
    intValues[IntConstantIDs::IMG_RES_Y] = resolution.y;
    intValues[IntConstantIDs::GRID_SIZE] = colsRows.x*colsRows.y;
    intValues[IntConstantIDs::COLS] = colsRows.x;
    intValues[IntConstantIDs::ROWS] = colsRows.y;
    intValues[IntConstantIDs::DISTANCE_ORDER] = params.distanceOrder;
    intValues[IntConstantIDs::SCAN_METRIC] = params.metric;
    intValues[IntConstantIDs::FOCUS_METHOD] = params.method;
    intValues[IntConstantIDs::FOCUS_METHOD_PARAMETER] = params.methodParameter;
    intValues[IntConstantIDs::CLOSEST_VIEWS] = params.closestViews;
    intValues[IntConstantIDs::BLOCK_SAMPLING] = params.blockSampling;
    intValues[IntConstantIDs::YUV_DISTANCE] = params.YUVDistance;
    int range = (params.scanRange > 0) ? params.scanRange : resolution.x/2;    
    intValues[IntConstantIDs::SCAN_RANGE] = range;
    hipMemcpyToSymbol(HIP_SYMBOL(Kernels::Constants::intConstants), intValues.data(), intValues.size() * sizeof(int));
    
    std::vector<float> floatValues(FloatConstantIDs::FLOAT_CONSTANTS_COUNT);
    floatValues[FloatConstantIDs::SPACE] = params.space;
    hipMemcpyToSymbol(HIP_SYMBOL(Kernels::Constants::floatConstants), floatValues.data(), floatValues.size() * sizeof(float));

    std::vector<void*> dataPointers(DataPointersIDs::POINTERS_COUNT);
    dataPointers[DataPointersIDs::SURFACES] = reinterpret_cast<void*>(surfaceObjectsArr);
    dataPointers[DataPointersIDs::TEXTURES] = reinterpret_cast<void*>(textureObjectsArr);
    dataPointers[DataPointersIDs::WEIGHTS] = reinterpret_cast<void*>(weightsGPU);
    dataPointers[DataPointersIDs::CLOSEST_WEIGHTS] = reinterpret_cast<void*>(closestFramesWeightsGPU);
    dataPointers[DataPointersIDs::CLOSEST_COORDS] = reinterpret_cast<void*>(closestFramesCoordsLinearGPU);
    hipMemcpyToSymbol(HIP_SYMBOL(Kernels::Constants::dataPointers), dataPointers.data(), dataPointers.size() * sizeof(void*));
}

void Interpolator::loadGPUOffsets(glm::vec2 viewCoordinates)
{
    std::vector<float2> offsets(colsRows.x*colsRows.y);
    for(int col=0; col<colsRows.x; col++)
        for(int row=0; row<colsRows.y; row++)
        {
            int gridID = row*colsRows.x + col; 
            float2 offset{(col-viewCoordinates.x)/colsRows.x, (row-viewCoordinates.y)/colsRows.y};
            offsets[gridID] = offset;
        }
    hipMemcpyToSymbol(HIP_SYMBOL(Kernels::offsets), offsets.data(), offsets.size() * sizeof(float2));
}

std::vector<float> Interpolator::generateWeights(glm::vec2 coords)
{
    auto maxDistance = glm::distance(glm::vec2(0,0), glm::vec2(colsRows));
    float weightSum{0};
    std::vector<float> weightVals;
    for(int col=0; col<colsRows.x; col++)
        for(int row=0; row<colsRows.y; row++)
        {
            float weight = maxDistance - glm::distance(coords, glm::vec2(col, row));
            weightSum += weight;
            weightVals.push_back(weight);
        }
    for(auto &weight : weightVals)
        weight /= weightSum; 
    return weightVals;
}

void Interpolator::loadGPUWeights(glm::vec2 viewCoordinates)
{
    hipMalloc(reinterpret_cast<void **>(&weightsGPU), sizeof(float)*colsRows.x*colsRows.y);
    auto weights = generateWeights(viewCoordinates);
    hipMemcpy(weightsGPU, weights.data(), weights.size()*sizeof(float), hipMemcpyHostToDevice);
}

glm::vec2 Interpolator::InterpolationParams::parseCoordinates(std::string coordinates)
{
    constexpr char delim{'_'};
    std::vector <float> numbers;
    std::stringstream ss(coordinates); 
    std::string value; 
    while(getline(ss, value, delim))
        numbers.push_back(std::stof(value));

    return {numbers[0], numbers[1]};
}

void Interpolator::prepareClosestFrames(glm::vec2 viewCoordinates)
{
    constexpr int CLOSEST_FRAMES_COUNT{4};
    
    glm::ivec2 downCoords{glm::floor(viewCoordinates)};
    glm::ivec2 upCoords{glm::ceil(viewCoordinates)};
    
    std::vector<float> closestFramesWeights(CLOSEST_FRAMES_COUNT);
    std::vector<glm::ivec2> closestFramesCoords(CLOSEST_FRAMES_COUNT);
    glm::vec2 unitPos{glm::fract(viewCoordinates)};

    closestFramesCoords[ClosestFrames::TOP_LEFT] = {downCoords};
    closestFramesWeights[ClosestFrames::TOP_LEFT] = (1 - unitPos.x) * (1 - unitPos.y);
    
    closestFramesCoords[ClosestFrames::TOP_RIGHT] = {upCoords.x, downCoords.y};;
    closestFramesWeights[ClosestFrames::TOP_RIGHT] = unitPos.x * (1 - unitPos.y);
    
    closestFramesCoords[ClosestFrames::BOTTOM_LEFT] = {downCoords.x, upCoords.y};
    closestFramesWeights[ClosestFrames::BOTTOM_LEFT] = (1 - unitPos.x) * unitPos.y;

    closestFramesCoords[ClosestFrames::BOTTOM_RIGHT] = {upCoords};
    closestFramesWeights[ClosestFrames::BOTTOM_RIGHT] = unitPos.x * unitPos.y;
  
    std::vector<int> closestFramesCoordsLinear;
    for(auto const &coords : closestFramesCoords)
       closestFramesCoordsLinear.push_back(coords.y*colsRows.x+coords.x); 
     
    hipMalloc(reinterpret_cast<void **>(&closestFramesCoordsLinearGPU), sizeof(int)*closestFramesCoordsLinear.size());
    hipMemcpy(closestFramesCoordsLinearGPU, closestFramesCoordsLinear.data(), closestFramesCoordsLinear.size()*sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc(reinterpret_cast<void **>(&closestFramesWeightsGPU), sizeof(float)*closestFramesWeights.size());
    hipMemcpy(closestFramesWeightsGPU, closestFramesWeights.data(), closestFramesWeights.size()*sizeof(float), hipMemcpyHostToDevice);
}

ScanMetric Interpolator::InterpolationParams::parseMetric(std::string metric)
{
    if(metric == "VAR")
        return ScanMetric::VARIANCE;
    if(metric == "RANGE")
        return ScanMetric::RANGE;
    std::cerr << "Scan metric set to default." << std::endl;
    return ScanMetric::VARIANCE;
}
 
FocusMethod Interpolator::InterpolationParams::parseMethod(std::string method)
{
    if(method == "OD")
        return FocusMethod::ONE_DISTANCE;
    else if(method == "BF")
        return FocusMethod::BRUTE_FORCE;
    std::cerr << "Scan method set to default." << std::endl;
    return FocusMethod::BRUTE_FORCE;
} 

void Interpolator::interpolate(InterpolationParams params)
{
    glm::vec2 coords = glm::vec2(colsRows-1)*params.coordinates;
    loadGPUWeights(coords);
    prepareClosestFrames(coords);
    loadGPUOffsets(coords);   
    loadGPUConstants(params);
    
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(resolution.x/dimBlock.x, resolution.y/dimBlock.y, 1);

    std::cout << "Elapsed time: "<<std::endl;
    float avgTime{0};
    for(int i=0; i<params.runs; i++)
    {
        Timer timer;
        Kernels::process<<<dimGrid, dimBlock, sharedSize>>>();
        auto time = timer.stop();
        avgTime += time;
        std::cout << "Run #" << i<< ": " << time << " ms" << std::endl;
    }
    std::cout << "Average of " << params.runs << " runs: " << avgTime/params.runs << " ms" << std::endl;

    storeResults(params.outputPath);
}

void Interpolator::storeResults(std::string path)
{
    std::cout << "Storing results..." << std::endl;
    LoadingBar bar(OUTPUT_SURFACE_COUNT);
    std::vector<uint8_t> data(resolution.x*resolution.y*resolution.z, 255);

    for(int i=0; i<OUTPUT_SURFACE_COUNT; i++) 
    {
        hipMemcpy2DFromArray(data.data(), resolution.x*resolution.z, reinterpret_cast<hipArray*>(surfaceOutputArrays[i]), 0, 0, resolution.x*resolution.z, resolution.y, hipMemcpyDeviceToHost);
        stbi_write_png((std::filesystem::path(path)/(fileNames[i]+".png")).c_str(), resolution.x, resolution.y, resolution.z, data.data(), resolution.x*resolution.z);
        bar.add();
    }
}
