#define GLM_FORCE_SWIZZLE
#include <sstream>
#include <hip/hip_runtime.h>
#include "interpolator.h"
#include "kernels.cu"
#include "lfLoader.h"
#include "libs/loadingBar/loadingbar.hpp"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "libs/stb_image_write.h"

class Timer
{
    public:
    Timer()
    {    
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent);
    }
    float stop()
    {
        hipEventRecord(stopEvent);
        hipEventSynchronize(stopEvent);
        float time = 0;
        hipEventElapsedTime(&time, startEvent, stopEvent);
        hipEventDestroy(startEvent);
        hipEventDestroy(stopEvent);
        return time; 
    };
    private:
    hipEvent_t startEvent, stopEvent;
};

Interpolator::Interpolator(std::string inputPath) : input{inputPath}
{
    init();
}

Interpolator::~Interpolator()
{
    hipDeviceReset();
}

void Interpolator::init()
{
    loadGPUData();
    loadGPUConstants();
    sharedSize = sizeof(half)*colsRows.x*colsRows.y;
}

int Interpolator::createTextureObject(const uint8_t *data, glm::ivec3 size)
{
    hipChannelFormatDesc channels = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipArray *arr;
    hipMallocArray(&arr, &channels, size.x, size.y);
    hipMemcpy2DToArray(arr, 0, 0, data, size.x*size.z, size.x*size.z, size.y, hipMemcpyHostToDevice);
    
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = arr;
    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;
    hipTextureObject_t texObj{0};
    hipCreateTextureObject(&texObj, &texRes, &texDescr, NULL);
    return texObj;
}

std::pair<int, int*> Interpolator::createSurfaceObject(glm::ivec3 size, const uint8_t *data)
{
    auto arr = loadImageToArray(data, size);
    hipResourceDesc surfRes;
    memset(&surfRes, 0, sizeof(hipResourceDesc));
    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = reinterpret_cast<hipArray*>(arr);
    hipSurfaceObject_t surfObj = 0;
    hipCreateSurfaceObject(&surfObj, &surfRes);
    return {surfObj, arr};
}

int* Interpolator::loadImageToArray(const uint8_t *data, glm::ivec3 size)
{
    hipChannelFormatDesc channels = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned); 
    hipArray *arr;
    hipMallocArray(&arr, &channels, size.x, size.y, hipArraySurfaceLoadStore);
    if(data != nullptr)
        hipMemcpy2DToArray(arr, 0, 0, data, size.x*size.z, size.x*size.z, size.y, hipMemcpyHostToDevice);
    return reinterpret_cast<int*>(arr);
}

void Interpolator::loadGPUData()
{
    LfLoader lfLoader;
    lfLoader.loadData(input);
    colsRows = lfLoader.getColsRows();
    resolution = lfLoader.imageResolution();

    std::cout << "Uploading data to GPU..." << std::endl;
    LoadingBar bar(lfLoader.imageCount()+OUTPUT_SURFACE_COUNT);

    /*
    std::vector<hipTextureObject_t> textures;
    for(int col=0; col<colsRows.x; col++)
        for(int row=0; row<colsRows.y; row++)
        { 
            textures.push_back(createTextureObject(lfLoader.image({col, row}).data(), resolution)); 
            bar.add();
        }

    hipMalloc(&textureObjectsArr, textures.size()*sizeof(hipTextureObject_t));
    hipMemcpy(textureObjectsArr, textures.data(), textures.size()*sizeof(hipTextureObject_t), hipMemcpyHostToDevice);
    */

    
    std::vector<hipSurfaceObject_t> surfaces;
    for(int col=0; col<colsRows.x; col++)
        for(int row=0; row<colsRows.y; row++)
        {
            auto surface = createSurfaceObject(resolution, lfLoader.image({col, row}).data());
            surfaces.push_back(surface.first);  
            surfaceInputArrays.push_back(surface.second);
            bar.add();
        }

    for(int i=0; i<OUTPUT_SURFACE_COUNT; i++)
    {
        auto surface = createSurfaceObject(resolution);
        surfaces.push_back(surface.first);  
        surfaceOutputArrays.push_back(surface.second);
        bar.add();
    }
    hipMalloc(&surfaceObjectsArr, surfaces.size()*sizeof(hipTextureObject_t));
    hipMemcpy(surfaceObjectsArr, surfaces.data(), surfaces.size()*sizeof(hipSurfaceObject_t), hipMemcpyHostToDevice);
}

void Interpolator::loadGPUConstants()
{
    std::vector<int> values{resolution.x, resolution.y, colsRows.x, colsRows.y, colsRows.x*colsRows.y, FileNames::FOCUS_MAP, FileNames::RENDER_IMAGE};
    hipMemcpyToSymbol(HIP_SYMBOL(Kernels::constants), values.data(), values.size() * sizeof(int));
}

std::vector<float> Interpolator::generateWeights(glm::vec2 coords)
{
    auto maxDistance = glm::distance(glm::vec2(0,0), glm::vec2(colsRows));
    float weightSum{0};
    std::vector<float> weightVals;
    for(int col=0; col<colsRows.x; col++)
        for(int row=0; row<colsRows.y; row++)
        {
            float weight = maxDistance - glm::distance(coords, glm::vec2(col, row));
            weightSum += weight;
            weightVals.push_back(weight);
        }
    for(auto &weight : weightVals)
        weight /= weightSum; 
    return weightVals;
}

void Interpolator::loadGPUWeights(glm::vec2 viewCoordinates)
{
    hipMalloc(reinterpret_cast<void **>(&weightsGPU), sizeof(half)*colsRows.x*colsRows.y);
    std::vector<half> weights;
    auto weightsFloat = generateWeights(viewCoordinates);
    for(const auto & w : weightsFloat)
            weights.push_back(static_cast<half>(w));
    hipMemcpy(weightsGPU, weights.data(), weights.size()*sizeof(half), hipMemcpyHostToDevice);
}

Interpolator::Method Interpolator::parseMethod(std::string method)
{
    if(method == "BF")
        return BRUTE_FORCE;
    return BRUTE_FORCE; 
}

glm::vec2 Interpolator::parseCoordinates(std::string coordinates)
{
    constexpr char delim{'_'};
    std::vector <std::string> numbers;
    std::stringstream ss(coordinates); 
    std::string value; 
    while(getline(ss, value, delim))
        numbers.push_back(value);

    glm::vec2 coords;
    int i{0};
    for (const auto &number : numbers)
    {
        float value = std::stof(number);
        coords[i] = value*colsRows[i%2];
        i++;
    }
    return coords;
}

void Interpolator::interpolate(std::string outputPath, std::string coordinates, std::string method)
{
    glm::vec2 coords = parseCoordinates(coordinates);
    loadGPUWeights(coords);
    
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(resolution.x/dimBlock.x, resolution.y/dimBlock.y, 1);

    Timer timer;
    std::cout << "Elapsed time: "<<std::endl;
    Kernels::process<<<dimGrid, dimBlock, sharedSize>>>(reinterpret_cast<hipTextureObject_t*>(textureObjectsArr), reinterpret_cast<hipSurfaceObject_t*>(surfaceObjectsArr), reinterpret_cast<half*>(weightsGPU));
    std::cout << timer.stop() << " ms" << std::endl;

    storeResults(outputPath);
}

void Interpolator::storeResults(std::string path)
{
    std::cout << "Storing results..." << std::endl;
    LoadingBar bar(OUTPUT_SURFACE_COUNT);
    std::vector<uint8_t> data(resolution.x*resolution.y*resolution.z, 255);

    for(int i=0; i<OUTPUT_SURFACE_COUNT; i++) 
    {
        hipMemcpy2DFromArray(data.data(), resolution.x*resolution.z, reinterpret_cast<hipArray*>(surfaceOutputArrays[i]), 0, 0, resolution.x*resolution.z, resolution.y, hipMemcpyDeviceToHost);
        stbi_write_png((std::filesystem::path(path)/(fileNames[i]+".png")).c_str(), resolution.x, resolution.y, resolution.z, data.data(), resolution.x*resolution.z);
        bar.add();
    }
}
