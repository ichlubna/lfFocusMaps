#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <hip/hip_fp16.h>
#include <hiprand/hiprand_kernel.h>
#include "methods.h"

namespace Kernels
{
    constexpr int BLOCK_SAMPLE_COUNT{5};
    constexpr int PIXEL_SAMPLE_COUNT{1};
    constexpr int CLOSEST_COUNT{4}; 

    namespace Constants
    {
        enum TextureMode{NORMAL, MIP, SECONDARY};
        __device__ TextureMode textureMode{NORMAL};
        __device__ void setNormalTextures(){textureMode = NORMAL;};
        __device__ void setMipTextures(){textureMode = MIP;};
        __device__ void setSecondaryTextures(){textureMode = SECONDARY;};

        __constant__ int intConstants[IntConstantIDs::INT_CONSTANTS_COUNT];
        __device__ int2 imgRes(){return {intConstants[IntConstantIDs::IMG_RES_X], intConstants[IntConstantIDs::IMG_RES_Y]};}
        __device__ int2 colsRows(){return{intConstants[IntConstantIDs::COLS], intConstants[IntConstantIDs::ROWS]};}
        __device__ int gridSize(){return intConstants[IntConstantIDs::GRID_SIZE];}
        __device__ int distanceOrder(){return intConstants[IntConstantIDs::DISTANCE_ORDER];}
        __device__ ScanMetric scanMetric(){return static_cast<ScanMetric>(intConstants[IntConstantIDs::SCAN_METRIC]);}
        __device__ FocusMethod focusMethod(){return static_cast<FocusMethod>(intConstants[IntConstantIDs::FOCUS_METHOD]);}
        __device__ bool closestViews(){return intConstants[IntConstantIDs::CLOSEST_VIEWS];}
        __device__ bool blockSampling(){return intConstants[IntConstantIDs::BLOCK_SAMPLING];}
        __device__ ColorDistance YUVDistance(){return static_cast<ColorDistance>(intConstants[IntConstantIDs::YUV_DISTANCE]);}
        __device__ bool blendAddressMode(){return intConstants[IntConstantIDs::BLEND_ADDRESS_MODE];}
        __device__ bool noMap(){return intConstants[IntConstantIDs::NO_MAP];}
        __device__ int ClockSeed(){return intConstants[IntConstantIDs::CLOCK_SEED];}
        
        __constant__ void* dataPointers[DataPointersIDs::POINTERS_COUNT];
        __device__ hipSurfaceObject_t* surfaces(){return reinterpret_cast<hipSurfaceObject_t*>(dataPointers[DataPointersIDs::SURFACES]);}
        __device__ hipTextureObject_t* textures()
        {
            switch(textureMode)
            {
                case NORMAL:
                    return reinterpret_cast<hipTextureObject_t*>(dataPointers[DataPointersIDs::TEXTURES]);
                break;
                case MIP:
                    return reinterpret_cast<hipTextureObject_t*>(dataPointers[DataPointersIDs::MIP_TEXTURES]);
                break;
                case SECONDARY:
                    return reinterpret_cast<hipTextureObject_t*>(dataPointers[DataPointersIDs::SECONDARY_TEXTURES]);
                break;
            }
        }
        __device__ float* closestWeights(){return reinterpret_cast<float*>(dataPointers[DataPointersIDs::CLOSEST_WEIGHTS]);}
        __device__ float* weights(){return reinterpret_cast<float*>(dataPointers[DataPointersIDs::WEIGHTS]);}
        __device__ int* closestCoords(){return reinterpret_cast<int*>(dataPointers[DataPointersIDs::CLOSEST_COORDS]);}
        
        __constant__ float floatConstants[FloatConstantIDs::FLOAT_CONSTANTS_COUNT];
        __device__ float scanSpace(){return floatConstants[FloatConstantIDs::SPACE];}
        __device__ float descentStartStep(){return floatConstants[FloatConstantIDs::DESCENT_START_STEP];}
        __device__ float scanRange(){return floatConstants[FloatConstantIDs::SCAN_RANGE];}
        __device__ float focusMethodParameter(){return floatConstants[FloatConstantIDs::FOCUS_METHOD_PARAMETER];}

        __constant__ float descentStartPoints[DESCENT_START_POINTS];
        __constant__ float hierarchySteps[HIERARCHY_DIVISIONS];
        __constant__ int hierarchySamplings[HIERARCHY_DIVISIONS];
    }

    __device__ constexpr int MAX_IMAGES{256};
    __constant__ float2 offsets[MAX_IMAGES];
    //extern __shared__ half localMemory[];

     template <typename T>
        class PixelArray
        {
            public:
            __device__ PixelArray(){};
            __device__ PixelArray(uchar4 pixel) : channels{T(pixel.x), T(pixel.y), T(pixel.z), T(pixel.w)}{};
            __device__ PixelArray(float4 pixel) : channels{T(pixel.x), T(pixel.y), T(pixel.z), T(pixel.w)}{};
            T channels[CHANNEL_COUNT]{0,0,0,0};
            __device__ T& operator[](int index){return channels[index];}
          
             __device__ uchar4 uch4() 
            {
                uchar4 result;
                auto data = reinterpret_cast<unsigned char*>(&result);
                for(int i=0; i<CHANNEL_COUNT; i++)
                    data[i] = __float2int_rn(channels[i]);
                return result;
            }
           
            __device__ void addWeighted(T weight, PixelArray<T> &value) 
            {    
                for(int j=0; j<CHANNEL_COUNT; j++)
                    //channels[j] += value[j]*weight;
                    channels[j] = __fmaf_rn(value[j], weight, channels[j]);
            }
            
            __device__ PixelArray<T> operator/= (const T &value)
            {
                for(int j=0; j<CHANNEL_COUNT; j++)
                    this->channels[j] /= value;
                return *this;
            }
            
            __device__ PixelArray<T> operator+= (const PixelArray<T> &value)
            {
                for(int j=0; j<CHANNEL_COUNT; j++)
                    this->channels[j] += value.channels[j];
                return *this;
            }
             
            __device__ PixelArray<T> operator+ (const PixelArray &value)
            {
                for(int j=0; j<CHANNEL_COUNT; j++)
                    this->channels[j] += value.channels[j];
                return *this;
            }
            
            __device__ PixelArray<T> operator-(const PixelArray &value)
            {
                for(int j=0; j<CHANNEL_COUNT; j++)
                    this->channels[j] -= value.channels[j];
                return *this;
            }
            
            __device__ PixelArray<T> operator/(const T &value)
            {
                for(int j=0; j<CHANNEL_COUNT; j++)
                    this->channels[j] /= value;
                return *this;
            }
            
            __device__ PixelArray<T> operator*(const T &value)
            {
                for(int j=0; j<CHANNEL_COUNT; j++)
                    this->channels[j] *= value;
                return *this;
            }
        };

    __device__ bool coordsOutside(int2 coords)
    {
        int2 resolution = Constants::imgRes();
        return (coords.x >= resolution.x || coords.y >= resolution.y);
    }

    __device__ int2 getImgCoords()
    {
        int2 coords;
        coords.x = (threadIdx.x + blockIdx.x * blockDim.x);
        coords.y = (threadIdx.y + blockIdx.y * blockDim.y);
        return coords;
    }
   
    namespace Pixel
    {
        //source: https://learn.microsoft.com/en-us/windows/win32/medfound/recommended-8-bit-yuv-formats-for-video-rendering
        template <typename T>        
        __device__ uchar4 RGBtoYUV(PixelArray<T> &rgb)
        {
            uchar4 yuv;
            yuv.x = ( __float2uint_rn(  66 * rgb[0] + 129 * rgb[1] +  25 * rgb[2] + 128) >> 8) +  16;
            yuv.y = ( __float2uint_rn( -38 * rgb[0] -  74 * rgb[1] + 112 * rgb[2] + 128) >> 8) + 128;
            yuv.z = ( __float2uint_rn( 112 * rgb[0] -  94 * rgb[1] -  18 * rgb[2] + 128) >> 8) + 128;
            return yuv;
        }
        
        template <typename T>        
        __device__ unsigned char RGBtoY(PixelArray<T> &rgb)
        {
            return (__float2uint_rn(  66 * rgb[0] + 129 * rgb[1] +  25 * rgb[2] + 128) >> 8) +  16;
        }


        template <typename T>
        __device__ float distance(PixelArray<T> &a, PixelArray<T> &b)
        {
            float dist{0};
            switch(Constants::YUVDistance())
            {
                case RGB:
                {
                    dist = max(max(abs(a[0]-b[0]), abs(a[1]-b[1])), abs(a[2]-b[2]));
                }
                break;
                
                case YUV:            
                {
                    auto yuvA = RGBtoYUV<T>(a);
                    auto yuvB = RGBtoYUV<T>(b);
                    dist = max(max(abs(yuvA.x-yuvB.x), abs(yuvA.y-yuvB.y)), abs(yuvA.z-yuvB.z));
                }
                break;
                
                case YUVw:            
                {
                    auto yuvA = RGBtoYUV(a);
                    auto yuvB = RGBtoYUV(b);
                    dist = max(max(abs(yuvA.x-yuvB.x)>>2, abs(yuvA.y-yuvB.y)), abs(yuvA.z-yuvB.z));
                }
                break;

                case Y:
                    dist = abs(RGBtoY<T>(a)-RGBtoY<T>(b));
                break;
            }
            return __powf (dist, Constants::distanceOrder());
        }

        __device__ void store(uchar4 px, int imageID, int2 coords)
        {
            surf2Dwrite<uchar4>(px, Constants::surfaces()[imageID], coords.x*sizeof(uchar4), coords.y);
        }

        template <typename T>
        __device__ PixelArray<T> load(int imageID, float2 coords)
        {
            int id = Constants::textures()[imageID];
            if(Constants::blendAddressMode())
            {
                constexpr float SPREAD{0.0015f};
                float offset{0};
                if(coords.x > 1.0 || coords.x < 0.0)
                    offset += floor(coords.x)*SPREAD;
                if(coords.y > 1.0 || coords.y < 0.0)
                    offset += floor(coords.y)*SPREAD;
                PixelArray<T> pixel;
                const float2 offsets[4] = {{offset, offset}, {offset, -offset}, {-offset, -offset}, {-offset, offset}};
                for(int i=0; i<4; i++)
                    pixel += PixelArray<T>{tex2D<float4>(id, coords.x+offsets[i].x, coords.y+offsets[i].y)}; 
                return (pixel/4)*UCHAR_MAX;
            }    
            else
                return PixelArray<T>{tex2D<float4>(id, coords.x, coords.y)}*UCHAR_MAX;
        } 
    }

    namespace ScanMetrics
    {
        template <typename T>
        class OnlineVariance
        {
            private:
            float n{0};
            PixelArray<T> m;
            float m2{0};
            
            public:
            __device__ void add(PixelArray<T> val)
            {
               float distance = Pixel::distance<T>(m, val);
               n++;
               PixelArray<T> delta = val-m;
               m += delta/static_cast<T>(n);
               //m2 += distance * Pixel::distance(m, val);
               m2 = __fmaf_rn(distance, Pixel::distance(m, val), m2);

            }
            __device__ float dispersionAmount()
            {
                return m2/(n-1);    
            }      
            __device__ OnlineVariance& operator+=(const PixelArray<T>& rhs){

              add(rhs);
              return *this;
            }
        };
        
        template <typename T>
        class Range
        {
            private:
            PixelArray<T> minCol{float4{FLT_MAX, FLT_MAX, FLT_MAX, FLT_MAX}};
            PixelArray<T> maxCol{float4{FLT_MIN, FLT_MIN, FLT_MIN, FLT_MIN}};
            
            public:
            __device__ void add(PixelArray<T> val)
            {
                minCol[0] = min(minCol[0],val[0]);
                minCol[1] = min(minCol[1],val[1]);
                minCol[2] = min(minCol[2],val[2]);
                maxCol[0] = max(maxCol[0],val[0]);
                maxCol[1] = max(maxCol[1],val[1]);
                maxCol[2] = max(maxCol[2],val[2]);
            }
            __device__ float dispersionAmount()
            {
                return Pixel::distance(minCol, maxCol);  
            }      
            __device__ Range& operator+=(const PixelArray<T>& rhs){

              add(rhs);
              return *this;
            }
        };
       
        template <typename T>
        class Percentile
        {
            private:
            float stepUp;
            float stepDown;
            float step{10};
            float value{0};
            bool init{false};

            public:
            __device__ Percentile(float percentile)
            {
                stepUp = 1.0f-percentile;
                stepDown = percentile;
            }
            __device__ void add(PixelArray<T> pixel)
            {
                PixelArray<T> origin;
                float dist = Pixel::distance<T>(pixel, origin);

                if(!init)
                {
                    value = dist;
                    step = max(dist, 1.0f);
                    init = true;
                    return;
                }
                if(value > dist)
                    value -= step*stepUp;
                else if(value < dist)
                    value += step*stepDown;
                if(abs(value-dist) < step)
                   step /= 2.0f; 
            }
            __device__ float result()
            {
                return value;
            }
        };
 
        template <typename T>
        class IQR
        {
            private:
            Percentile<T> first{0.25};            
            Percentile<T> second{0.75};            
 
            public:
            __device__ void add(PixelArray<T> val)
            {
                first.add(val); 
                second.add(val); 
            }
            __device__ float dispersionAmount()
            {
                return second.result()-first.result();
            }      
            __device__ IQR& operator+=(const PixelArray<T>& rhs){

              add(rhs);
              return *this;
            }
        };
      
        template <typename T>
        class Mad
        {
            private:
            PixelArray<T> last;
            PixelArray<T> sample;
            float dist{0};
            int n{0};
            static constexpr int SAMPLE_CYCLE{5};
             
            public:
            __device__ void add(PixelArray<T> val)
            { 
                if(n%SAMPLE_CYCLE == 0)
                    sample = val;
                if(n != 0)
                {
                    dist += Pixel::distance(val, sample);
                    dist += Pixel::distance(val, last); 
                }
                last = val; 
                n++;
            }
            __device__ float dispersionAmount()
            {
                return dist;
            }      
            __device__ Mad& operator+=(const PixelArray<T>& rhs){

              add(rhs);
              return *this;
            }
        };
    }

    template<typename T>
    __device__ float2 focusCoords(int gridID, T pxCoords, float focus)
    {
        float2 offset = offsets[gridID];
        float2 coords{pxCoords.x-offset.x*focus, pxCoords.y-offset.y*focus};
        return coords;
    }

    __device__ float transformFocus(float focus, int range, float space)
    {
        if(space != 1.0f)
        {
            float normalized = focus/range;
            return __powf(normalized, space)*range;
        }
        return focus;
    }

    namespace FocusLevel
    {      
        template<int blockSize, typename T> 
        __device__ void evaluateBlock(int gridID, float focus, float2 coords, T *dispersions)
        {
            float transformedFocus = transformFocus(focus, Constants::scanRange(), Constants::scanSpace());
            const float2 BLOCK_OFFSETS[]{ {0,0}, {-1,0.5}, {0.5, 1}, {1,-0.5}, {-0.5,-1} };
            for(int blockPx=0; blockPx<blockSize; blockPx++)
            {
                float2 inBlockCoords{coords.x+BLOCK_OFFSETS[blockPx].x, coords.y+BLOCK_OFFSETS[blockPx].y};
                auto px{Pixel::load<float>(gridID, focusCoords(gridID, inBlockCoords, transformedFocus))};
                dispersions[blockPx] += px;
            }
        }

        template<typename T, int blockSize, bool closest=false>
        __device__ float evaluateDispersion(float2 coords, float focus)
        {
            auto cr = Constants::colsRows();
            T dispersionCalc[blockSize];
                
            int gridID = 0;

            if constexpr (closest)
            {  
                auto closestCoords = Constants::closestCoords();
                for(int i=0; i<CLOSEST_COUNT; i++) 
                {     
                    int gridID = closestCoords[i];
                    evaluateBlock<blockSize>(gridID, focus, coords, dispersionCalc);
                }           
            }
            else
                for(int row=0; row<cr.y; row++) 
                {     
                    gridID = row*cr.x;
                    for(int col=0; col<cr.x; col++) 
                    {
                        evaluateBlock<blockSize>(gridID, focus, coords, dispersionCalc);
                        gridID++;
                    }
                }

            float finalDispersion{0};
            for(int blockPx=0; blockPx<blockSize; blockPx++)
                finalDispersion += dispersionCalc[blockPx].dispersionAmount();
            return finalDispersion;
        }

        template<int blockSize, bool closest, typename...TAIL>
        __device__ typename std::enable_if_t<sizeof...(TAIL)==0, float> 
        call(int,ScanMetric,float2,float){}

        template<int blockSize, bool closest, typename H, typename...TAIL>
        __device__ float call(int n,ScanMetric type, float2 coords, float focus)
        {
            if(n==type)
                return evaluateDispersion<H, blockSize, closest>(coords, focus);
            return call<blockSize, closest, TAIL...>(n+1,type, coords, focus);
        }

        template<int blockSize, bool closest=false>
        __device__ float dispersion(ScanMetric t, float2 coords, float focus)
        {
            return call<blockSize, closest, ScanMetrics::OnlineVariance<float>, ScanMetrics::Range<float>, ScanMetrics::IQR<float>, ScanMetrics::Mad<float>>(0,t, coords, focus);
        }

        __device__ float evaluate(float2 coords, float focus)
        {
            auto closestViews = Constants::closestViews(); 
            auto blockSampling = Constants::blockSampling();
            auto scanMetric = Constants::scanMetric();
 
            if(closestViews)
                if(blockSampling)
                    return dispersion<BLOCK_SAMPLE_COUNT, true>(scanMetric, coords, focus);
                else
                    return dispersion<PIXEL_SAMPLE_COUNT, true>(scanMetric, coords, focus);
            else
                if(blockSampling)
                    return dispersion<BLOCK_SAMPLE_COUNT>(scanMetric, coords, focus);
                else
                    return dispersion<PIXEL_SAMPLE_COUNT>(scanMetric, coords, focus);
        }
       
        template<bool closest=false>
        __device__ uchar4 render(float2 coords, float focus)
        {
            auto cr = Constants::colsRows();
            PixelArray<float> sum;
            int gridID = 0; 
            
            if constexpr (closest)
            {
                auto closestCoords = Constants::closestCoords();
                auto weights = Constants::closestWeights();
                for(int i=0; i<CLOSEST_COUNT; i++) 
                {
                    gridID = closestCoords[i];
                    auto px{Pixel::load<float>(gridID, focusCoords(gridID, coords, focus))};
                    sum.addWeighted(weights[i], px);
                }
            }
            else
            {
                auto weights = Constants::weights();
                for(int row=0; row<cr.y; row++) 
                {     
                    gridID = row*cr.x;
                    for(int col=0; col<cr.x; col++) 
                    {
                        auto px{Pixel::load<float>(gridID, focusCoords(gridID, coords, focus))};
                        sum.addWeighted(weights[gridID], px);
                        gridID++;
                    }
                }
            }
            return sum.uch4();
        }      
    }
    
    namespace Focusing
    {    
        class Optimum
        {
            public:
            float optimalFocus{0};
            float minDispersion{FLT_MAX};
            __device__ bool add(float focus, float dispersion)
            {
                if(dispersion < minDispersion)
                {
                   minDispersion = dispersion;
                   optimalFocus = focus; 
                   return true;
                }
                return false;
            }
            __device__ void addForce(float focus, float dispersion)
            {
                   minDispersion = dispersion;
                   optimalFocus = focus; 
            }
        }; 

        __device__ Optimum& minOpt(Optimum &a, Optimum &b)
        {
            if(a.minDispersion < b.minDispersion)
                return a;
            else
                return b;
        }

        __device__ float bruteForce(float2 coords)
        {
            int steps = Constants::focusMethodParameter();
            float stepSize{static_cast<float>(Constants::scanRange())/steps};
            float focus{0.0f};
            Optimum optimum;
            
            for(int step=0; step<steps; step++)
            {
                float dispersion = FocusLevel::evaluate(coords, focus);
                optimum.add(focus, dispersion);
                focus += stepSize;  
            }
            return optimum.optimalFocus;
        }
        
        __device__ float randomSampling(float2 coords)
        {
            unsigned int linearID = coords.y*Constants::imgRes().x + coords.x;
            hiprandState state;
            hiprand_init(Constants::ClockSeed()+linearID, 0, 0, &state);
            int steps = Constants::focusMethodParameter();
            int range = Constants::scanRange();
            Optimum optimum;
            
            for(int step=0; step<steps; step++)
            {
                float focus = range*hiprand_uniform(&state) ;
                float dispersion = FocusLevel::evaluate(coords, focus);
                optimum.add(focus, dispersion);
            }

            return optimum.optimalFocus;
        }
       
        template <bool stochastic> 
        __device__ float hierarchical(float2 coords)
        {
            hiprandState state;
            if constexpr (stochastic)
            {
                unsigned int linearID = coords.y*Constants::imgRes().x + coords.x;
                hiprand_init(Constants::ClockSeed()+linearID, 0, 0, &state);
            }

            int range = Constants::scanRange();
            Optimum optimum; 
            bool divide{true};
            float2 dividedRange{0, static_cast<float>(range)};
            for(int d=0; d<HIERARCHY_DIVISIONS; d++)
            {  
                Optimum leftRightOptimum[2];
                int sampling = Constants::hierarchySamplings[d];
                int samplingHalf = sampling/2;
                float focus = dividedRange.x;
                for(int i=0; i<sampling; i++)
                {
                    float disp = FocusLevel::evaluate(coords, focus);
                    leftRightOptimum[(i<samplingHalf) ? 0 : 1].add(focus, disp);
                    focus += Constants::hierarchySteps[d];
                }

                if(leftRightOptimum[0].minDispersion < leftRightOptimum[1].minDispersion)
                {
                    divide = optimum.add(leftRightOptimum[0].optimalFocus, leftRightOptimum[0].minDispersion);
                    dividedRange = {0, dividedRange.y*0.5f};
                }
                else
                {
                    divide = optimum.add(leftRightOptimum[1].optimalFocus, leftRightOptimum[1].minDispersion);
                    dividedRange = {dividedRange.y*0.5f, dividedRange.y};
                }
                if(!divide)
                    break;
            }
            return optimum.optimalFocus;
        }
        
        template <bool stochastic> 
        __device__ float descent(float2 coords)
        {
            hiprandState state;
            if constexpr (stochastic)
            {
                unsigned int linearID = coords.y*Constants::imgRes().x + coords.x;
                hiprand_init(Constants::ClockSeed()+linearID, 0, 0, &state);
            }

            constexpr int MAX_STEPS{100};
            Optimum optimum[DESCENT_START_POINTS];
            constexpr float LEARN_RATE{0.1f};
            constexpr float MIN_STEP{0.5f};
            
            for(int p=0; p<DESCENT_START_POINTS; p++)
            {
                float focus = Constants::descentStartPoints[p];
                float step{Constants::descentStartStep()};
                for(int i=0; i<MAX_STEPS; i++)
                {      
                    float2 focusPair{focus-step, focus+step};
                    float2 dispersionPair{FocusLevel::evaluate(coords, focusPair.x), FocusLevel::evaluate(coords, focusPair.y)};
                    if(dispersionPair.x < dispersionPair.y)
                        optimum[p].addForce(focusPair.x, dispersionPair.x);
                    else
                        optimum[p].addForce(focusPair.y, dispersionPair.y);
                    step = LEARN_RATE*abs(focus-optimum[p].optimalFocus);
                    focus = optimum[p].optimalFocus;
                    if(step < MIN_STEP)
                        break;
                }
            }
            Optimum &minimal = optimum[0];
            for(int i=1; i<DESCENT_START_POINTS; i++)
                minimal = minOpt(minimal, optimum[i]);
            return minimal.optimalFocus;
        }
        
        template <bool stochastic> 
        __device__ float pyramid(float2 coords)
        {
            hiprandState state;
            if constexpr (stochastic)
            {
                unsigned int linearID = coords.y*Constants::imgRes().x + coords.x;
                hiprand_init(Constants::ClockSeed()+linearID, 0, 0, &state);
            }
        }
    }

    __global__ void process()
    {
        int2 threadCoords = getImgCoords(); 
        if(coordsOutside(threadCoords))
            return;
        float2 coords = {static_cast<float>(threadCoords.x)/Constants::imgRes().x,
                        static_cast<float>(threadCoords.y)/Constants::imgRes().y};

        //MemoryPartitioner<half> memoryPartitioner(localMemory);
        //auto localWeights = memoryPartitioner.array(gridSize());
        //loadWeightsSync<half>(weights, localWeights.data, gridSize()/2);

        float focus{0};
        Constants::setSecondaryTextures();
        switch(Constants::focusMethod())
        {
            case ONE_DISTANCE:
                focus = Constants::focusMethodParameter();
            break;

            case BRUTE_FORCE:
                focus = Focusing::bruteForce(coords);
            break;
            
            case RANDOM:
                focus = Focusing::randomSampling(coords);
            break;
            
            case PYRAMID:
                if(Constants::focusMethodParameter())
                    focus = Focusing::pyramid<true>(coords);
                else
                    focus = Focusing::pyramid<false>(coords);
            break;
            
            case HIERARCHY:
                if(Constants::focusMethodParameter())
                    focus = Focusing::hierarchical<true>(coords);
                else
                    focus = Focusing::hierarchical<false>(coords);
            break;
            
            case DESCENT:
                if(Constants::focusMethodParameter())
                    focus = Focusing::descent<true>(coords);
                else
                    focus = Focusing::descent<false>(coords);
            break;

            default:
            break;
        }       

        Constants::setNormalTextures(); 
        uchar4 color{0};
        if(Constants::closestViews())
            color = FocusLevel::render<true>(coords, focus);
        else
            color = FocusLevel::render(coords, focus);

        if(!Constants::noMap())
        {
            unsigned char focusColor = (focus/Constants::scanRange())*UCHAR_MAX;
            Pixel::store(uchar4{focusColor, focusColor, focusColor, UCHAR_MAX}, FileNames::FOCUS_MAP, threadCoords);
        }
        Pixel::store(color, FileNames::RENDER_IMAGE, threadCoords);
    }
}
